#include "hip/hip_runtime.h"
#include "MetaballCudaMgr.cuh"

#include <cassert>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include "./trunk/OgreMetaballs/MarchingCubesInterface.h"
//#include "../SPHSimLib/cuPrintf.cu"

__global__ void ScalarSphere(const float* position, 
					   const float radiusSquared,
					   const float* gridVertex, float* scalar,
					   const int ParticleCount, const int GridVertexCount);

MetaballCudaMgr* MetaballCudaMgr::m_singleton=NULL;

MetaballCudaMgr::MetaballCudaMgr()
{
//	cudaPrintfInit();
	d_spherePosition = NULL;
	d_spherePositionsCount = 0;
	sphereRadius = 1.0;

	d_samplingGridVertices = NULL;
	d_samplingGridVerticesCount = 0;
	
	d_samplingGridVerticesScalar = NULL;
	d_samplingGridVerticesScalarCount = 0;
	h_samplingGridVerticesScalar = NULL;
}
MetaballCudaMgr::~MetaballCudaMgr()
{
//	cudaPrintfEnd();
}
MetaballCudaMgr* MetaballCudaMgr::getSingletonPtr()
{
	if(!m_singleton)
	{
		m_singleton = new MetaballCudaMgr();
	}
	return m_singleton;
}
//
void MetaballCudaMgr::mallocSpherePosition(const std::size_t elementCount, const std::size_t elementSize)
{
	assert( elementSize==sizeof(float)*3 );
	d_spherePositionsCount = elementCount;
	CUDA_SAFE_CALL( hipMalloc((void**)&d_spherePosition, elementCount*elementSize) );
}
void MetaballCudaMgr::setSpherePosition(const float *src, const std::size_t elementCount)
{
	assert(elementCount<=d_spherePositionsCount);
	CUDA_SAFE_CALL( hipMemcpy(d_spherePosition, src, d_spherePositionsCount*sizeof(float)*3, hipMemcpyHostToDevice) );
	d_spherePositionsCount = elementCount;
}
void MetaballCudaMgr::freeSpherePosition()
{
	if(d_spherePosition)
	{
		CUDA_SAFE_CALL( hipFree(d_spherePosition) );
		d_spherePositionsCount = 0;
	}
}
//
void MetaballCudaMgr::mallocSamplingGridVertices(const std::size_t elementCount, const std::size_t elementSize)
{
	assert( elementSize==sizeof(float)*4 );
	d_samplingGridVerticesCount = elementCount;
	CUDA_SAFE_CALL( hipMalloc((void**)&d_samplingGridVertices, elementCount*elementSize) );
	//CUDA_SAFE_CALL( hipMalloc((void**)&d_samplingGridVerticesScalar,   elementCount*sizeof(float)) );
}
void MetaballCudaMgr::setSamplingGridVertices(const float *src, const std::size_t elementCount)
{
	assert(elementCount<=d_samplingGridVerticesCount);
	CUDA_SAFE_CALL( hipMemcpy(d_samplingGridVertices, src, d_samplingGridVerticesCount*sizeof(float)*3, hipMemcpyHostToDevice) );
	d_samplingGridVerticesCount = elementCount;
}
void MetaballCudaMgr::freeSamplingGridVertices()
{
	if(d_samplingGridVertices)
	{
		CUDA_SAFE_CALL( hipFree(d_samplingGridVertices) );
		d_samplingGridVerticesCount = 0;
	}
}
//
void MetaballCudaMgr::mallocSamplingGridVerticesScalar(const std::size_t elementCount, const std::size_t elementSize)
{
	assert( elementSize==sizeof(float) );
	d_samplingGridVerticesScalarCount = elementCount;
	assert( d_samplingGridVerticesScalarCount == d_samplingGridVerticesCount);
	CUDA_SAFE_CALL( hipMalloc((void**)&d_samplingGridVerticesScalar, elementCount*elementSize) );
	
	h_samplingGridVerticesScalar = (float*)malloc(elementCount*elementSize);
	assert(h_samplingGridVerticesScalar);
}
// void MetaballCudaMgr::setSamplingGridVerticesScalar(const float *src, const std::size_t elementCount)
// {
// 	assert(elementCount<=d_samplingGridVerticesScalarCount);
// 	CUDA_SAFE_CALL( hipMemcpy(d_samplingGridVerticesScalar, src, d_samplingGridVerticesScalarCount*sizeof(float)*3, hipMemcpyHostToDevice) );
// 	d_samplingGridVerticesScalarCount = elementCount;
// 	assert( d_samplingGridVerticesScalarCount == d_samplingGridVerticesCount);
// }
void MetaballCudaMgr::freeSamplingGridVerticesScalar()
{
	if(d_samplingGridVerticesScalar)
	{
		CUDA_SAFE_CALL( hipFree(d_samplingGridVerticesScalar) );
		d_samplingGridVerticesScalarCount = 0;
		
		free(h_samplingGridVerticesScalar);
	}
}
void MetaballCudaMgr::Scalar(const int gridSizeX, const int gridSizeY, const int gridSizeZ)
{
	//printf("MetaballCudaMgr::Scalar(%d, %d, %d)\n", gridSizeX, gridSizeY, gridSizeZ);
	//dim3 grid(gridSizeX, gridSizeY, gridSizeZ);
	int threadsPerBlock = 256;
	//assign a thread for each grid vertex
    int blocksPerGrid = (d_samplingGridVerticesCount + threadsPerBlock - 1) / threadsPerBlock;
	//printf("ScalarSphere<<<%d, %d>>>\n", blocksPerGrid, threadsPerBlock);

	ScalarSphere<<<blocksPerGrid, threadsPerBlock>>>(
		d_spherePosition, 
		sphereRadius*sphereRadius,
		d_samplingGridVertices, 
		d_samplingGridVerticesScalar,
		d_spherePositionsCount,
		d_samplingGridVerticesCount
	);
	//cudaPrintfDisplay(stdout, true);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    CUDA_SAFE_CALL( hipMemcpy(
		h_samplingGridVerticesScalar, 
		d_samplingGridVerticesScalar, 
		d_samplingGridVerticesScalarCount*sizeof(float), 
		hipMemcpyDeviceToHost
		) 
	);
}
void MetaballCudaMgr::ScalarCallback(SamplingGridVertice *vertex, const std::size_t vertexCount)
{
	assert(vertexCount==d_samplingGridVerticesScalarCount);
	for(size_t i=0; i<vertexCount; ++i)
	{
        vertex[i].ScalarValue = h_samplingGridVerticesScalar[i];
	}
}
//
__global__ void ScalarSphere(const float* position, 
					   const float radiusSquared,
					   const float* gridVertexPos, float* gridVertexScalar,
					   const int ParticleCount, const int GridVertexCount)
{
	uint blockId = blockIdx.y*gridDim.x + blockIdx.x;
    uint vi = blockId*blockDim.x + threadIdx.x;
#if __CUDA_ARCH__ >= 200
	//printf("grid<%d, %d, %d>, ",                       gridDim.x,                  gridDim.y,                 gridDim.z);
	//printf("block<%d/%d, %d/%d, %d/%d>, ",blockIdx.x, blockDim.x,     blockIdx.y, blockDim.y,    blockIdx.z, blockDim.z);
	//printf("thread<%d, %d, %d>\n",       threadIdx.x,                threadIdx.y,               threadIdx.z            );
#endif	
	uint pi = 0;
	
	float scalarSum = 0.0f;
	
	if(vi < GridVertexCount)//gridVertexPos[vi]={gvx, gvy, gvz}
	{
		float gvx = gridVertexPos[4*vi  ];
		float gvy = gridVertexPos[4*vi+1];
		float gvz = gridVertexPos[4*vi+2];

		for(pi=0; pi<ParticleCount; ++pi)//position[pi]={px, py, pz}
		{
			float px = position[3*pi  ];
			float py = position[3*pi+1];
			float pz = position[3*pi+2];

			scalarSum += radiusSquared /((px-gvx)*(px-gvx)+ 
										(py-gvy)*(py-gvy)+ 
										(pz-gvz)*(pz-gvz));
		}
	}
	gridVertexScalar[vi] = scalarSum;
}
