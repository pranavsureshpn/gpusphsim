#include "hip/hip_runtime.h"

#include <stdio.h>
#include <memory.h>
#include <cutil.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>


typedef unsigned int uint;

texture<float, 1, hipReadModeElementType> a_tex;
texture<float, 1, hipReadModeElementType> b_tex;

__global__ void testKernel(float* a, float* b)
{
	uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if(index > 100) index-=100;

	a[index] = 10;

	__threadfence();

	a[index] = tex1Dfetch(a_tex,index) + 1;

	volatile float x = tex1Dfetch(a_tex,index);
	x = tex1Dfetch(a_tex,index);
	x = tex1Dfetch(a_tex,index);
	x = tex1Dfetch(a_tex,index);


	__threadfence();
	__threadfence_block();
//	__threadfence_system();
	__syncthreads();

	//volatile float bv = a[index+1];
	b[index] = a[index+1];
	//b[index] = tex1Dfetch(a_tex,index+1);
}

void testKernel()
{
	float* da;
	float* db;
	hipMalloc((void**)&da, 100*sizeof(float));
	hipMalloc((void**)&db, 100*sizeof(float));
	hipMemset(da,0,100*sizeof(float));
	hipMemset(db,0,100*sizeof(float));

	float* ha;
	float* hb;
	hipHostMalloc((void**)&ha, 100*sizeof(float));
	hipHostMalloc((void**)&hb, 100*sizeof(float));
	memset(ha,0,100*sizeof(float));
	memset(hb,0,100*sizeof(float));


	hipBindTexture(0, a_tex, da, 100*sizeof(float));
	hipBindTexture(0, b_tex, db, 100*sizeof(float));

	testKernel<<<1,101>>>(da,db);

	hipUnbindTexture(a_tex);
	hipUnbindTexture(b_tex);

	hipMemcpy(ha,da,100*sizeof(float),hipMemcpyDeviceToHost);
	hipMemcpy(hb,db,100*sizeof(float),hipMemcpyDeviceToHost);

	return;
}


