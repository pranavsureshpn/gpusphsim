#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <string.h>
//#include "opengl.h"

#include "CudaHelpers.h"

#include <cuda_gl_interop.h>

extern "C" 
{




int PrintDevices(int deviceCount, int deviceSelected)
{
    hipError_t err = hipSuccess;

    hipDeviceProp_t deviceProperty;
    for (int currentDeviceId = 0; currentDeviceId < deviceCount; ++currentDeviceId)
    {
        memset(&deviceProperty, 0, sizeof(hipDeviceProp_t));
        err = hipGetDeviceProperties(&deviceProperty, currentDeviceId);

        printf("\ndevice name: %s", deviceProperty.name);
        if (currentDeviceId == deviceSelected)
        {
            printf("    <----- creating hipCtx_t on this");    
        }
        printf("\n");

        printf("device sharedMemPerBlock: %d \n", deviceProperty.sharedMemPerBlock);
        printf("device totalGlobalMem: %d \n", deviceProperty.totalGlobalMem);
        printf("device regsPerBlock: %d \n", deviceProperty.regsPerBlock);
        printf("device warpSize: %d \n", deviceProperty.warpSize);
        printf("device memPitch: %d \n", deviceProperty.memPitch);
        printf("device maxThreadsPerBlock: %d \n", deviceProperty.maxThreadsPerBlock);
        printf("device maxThreadsDim[0]: %d \n", deviceProperty.maxThreadsDim[0]);
        printf("device maxThreadsDim[1]: %d \n", deviceProperty.maxThreadsDim[1]);
        printf("device maxThreadsDim[2]: %d \n", deviceProperty.maxThreadsDim[2]);
        printf("device maxGridSize[0]: %d \n", deviceProperty.maxGridSize[0]);
        printf("device maxGridSize[1]: %d \n", deviceProperty.maxGridSize[1]);
        printf("device maxGridSize[2]: %d \n", deviceProperty.maxGridSize[2]);
        printf("device totalConstMem: %d \n", deviceProperty.totalConstMem);
        printf("device major: %d \n", deviceProperty.major);
        printf("device minor: %d \n", deviceProperty.minor);
        printf("device clockRate: %d \n", deviceProperty.clockRate);
        printf("device textureAlignment: %d \n", deviceProperty.textureAlignment);
        printf("device deviceOverlap: %d \n", deviceProperty.deviceOverlap);
        printf("device multiProcessorCount: %d \n", deviceProperty.multiProcessorCount);

        printf("\n");
    }

    return err;
}

void CUDA_Init(int dev) 
{
	hipError_t res;
	int count;
	hipDeviceProp_t p;
	
    res = hipGetDeviceCount(&count);
	if(res != hipSuccess)
	{
		CUDA_CheckError("hipGetDeviceCount failed");
	}
	
	//LOG("CUDA_Init: %d available devices \n", count);

    PrintDevices(count, dev);

    if(dev >= count || dev < 0)
	{
		dev = cutGetMaxGflopsDeviceId();		
	}
    
	//LOG("CUDA_Init: Using device %d\n", dev);
	
	res = hipSetDevice(dev);
	if(res != hipSuccess)
	{
		CUDA_CheckError("hipSetDevice failed");
	}
	else 
	{		
		//LOG("CUDA_Init: Successfull hipSetDevice\n", dev);
	}

	res = cudaGLSetGLDevice(dev);
	if(res != hipSuccess)
	{
		CUDA_CheckError("cudaGLSetGLDevice failed");
	}
	else 
	{		
		//LOG("CUDA_Init: Successfull cudaGLSetGLDevice\n", dev);
	}
}

hipError_t CUDA_GLMapBufferObject(void **devPtr, GLuint bufObj)
{
	return cudaGLMapBufferObject(devPtr, bufObj);
}

hipError_t CUDA_GLUnmapBufferObject(void **devPtr, GLuint bufObj)
{
	hipError_t err = cudaGLUnmapBufferObject(bufObj);
	*devPtr = 0;
	return err;
}

// hipError_t CUDA_CreateVBO(GLuint &vbo, uint size) 
// {
// 	vbo = CreateVBO(size);
// 	return CUDA_RegisterVBO(vbo);
// }
// 
// hipError_t CUDA_CreateDataVBO(GLuint &vbo, uint size, GLvoid *data) 
// {
// 	vbo = CreateVBO(size, data);
// 	return CUDA_RegisterVBO(vbo);
// }
// 
// hipError_t CUDA_FreeVBO(GLuint &vbo) 
// {
// 	hipError_t err = CUDA_UnregisterVBO(vbo);
// 	FreeVBO(vbo);
// 	return err;
// }

hipError_t CUDA_RegisterVBO(GLuint vbo) 
{
    return cudaGLRegisterBufferObject(vbo);
}

hipError_t CUDA_UnregisterVBO(GLuint vbo) 
{
    return cudaGLUnregisterBufferObject(vbo);
}

void CUDA_CheckError(const char *msg) {
    hipError_t err = hipGetLastError();
    if(hipSuccess != err) {
//		DIE("Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		/*
		FILE *fp = fopen("error.txt", "w");
        fprintf(fp, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		fclose(fp);
        exit(EXIT_FAILURE);
		*/
    }                         
}




} // extern "C"

