#include "hip/hip_runtime.h"
#ifndef __K_SimpleSPH_Step1_cu__
#define __K_SimpleSPH_Step1_cu__

#include "K_UniformGrid_Utils.cu"
#include "K_SPH_Kernels.cu"
#include "K_SPH_Common.cu"

class Step1
{
public:

	struct Data
	{
		float sum_density;

		SimpleSPHData dParticleDataSorted;
	};

	class Calc
	{
	public:

		static __device__ void PreCalc(Data &data, uint const &index_i)
		{
			// read particle data from sorted arrays
			data.sum_density = 0;
		}

		static __device__ void ForNeighbor(Data &data, uint const &index_i, uint const &index_j, float3 const &r, float const& rlen, float const &rlen_sq)
		{
			// the density sum using Wpoly6 kernel
			data.sum_density += SPH_Kernels::Wpoly6::Kernel_Variable(cPrecalcParams.smoothing_length_pow2, r, rlen_sq);	
		}

		static __device__ void PostCalc(Data &data, uint index_i)
		{
			// Compute the density field at the current particle,
			// Calculate the W smoothing function for this particle, mass and the poly6_grad_coeff has been moved outside the sum because they are constant.
			float density = max(1.0, cFluidParams.particle_mass * cPrecalcParams.kernel_poly6_coeff * data.sum_density);
			data.dParticleDataSorted.density[index_i]= density;

			// ideal gas equation of state (by Desbrun and Cani in "Smoothed particles: A new paradigm for animating highly deformable bodies")
			data.dParticleDataSorted.pressure[index_i] = cFluidParams.rest_pressure + cFluidParams.gas_stiffness * (density - cFluidParams.rest_density);
		}
	};
};


__global__ void K_SumStep1(uint				numParticles,
							   NeighborList		dNeighborList, 
							   SimpleSPHData	dParticleDataSorted,
							   GridData const	dGridData
							   )								
{
	// particle index	
	uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;		
	if (index >= numParticles) return;
	
	Step1::Data data;
	data.dParticleDataSorted = dParticleDataSorted;

	float3 position_i = make_float3(FETCH(dParticleDataSorted, position, index));

	// Do calculations on particles in neighboring cells
#ifdef SPHSIMLIB_USE_NEIGHBORLIST
	UniformGridUtils::IterateParticlesInNearbyCells<SPHNeighborCalc<Step1::Calc, Step1::Data>, Step1::Data>(data, index, position_i, dNeighborList);	
#else
	UniformGridUtils::IterateParticlesInNearbyCells<SPHNeighborCalc<Step1::Calc, Step1::Data>, Step1::Data>(data, index, position_i, dGridData);
#endif

}

#endif