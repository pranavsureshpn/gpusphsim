#include "hip/hip_runtime.h"
#include "MetaballCudaMgr.cuh"

#include <cassert>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include "./trunk/OgreMetaballs/MarchingCubesInterface.h"
//#include "../SPHSimLib/cuPrintf.cu"

//---------------------------------------------------------------------------
__constant__ __device__ uint3 NBRSamples;
__constant__ __device__ float3 SpaceResolution;
__constant__ __device__ uint3 ExtendCubes;
__constant__ __device__ float sphereRadius;
__constant__ __device__ float sphereRadiusSquared;
//---------------------------------------------------------------------------
__global__ void ScalarSphere1D(
					   const float* position, 
					   const float threshold,
					   const float* gridVertex, float* scalar,
					   const int ParticleCount, const int GridVertexCount);

__global__ void ScalarSphere3D(
					   const float* position, 
					   const float* gridVertex, float* scalar,
					   const int ParticleCount, const int GridVertexCount);

__global__ void ScalarSphere1D_EffectRange(
					   const float* position, 
					   const float* gridVertex, float* scalar,
					   const int ParticleCount, const int GridVertexCount);

MetaballCudaMgr* MetaballCudaMgr::m_singleton=NULL;

MetaballCudaMgr::MetaballCudaMgr()
{
//	cudaPrintfInit();
	d_spherePosition = NULL;
	d_spherePositionsCount = 0;

	d_samplingGridVertices = NULL;
	d_samplingGridVerticesCount = 0;
	
	d_samplingGridVerticesScalar = NULL;
	d_samplingGridVerticesScalarCount = 0;
	h_samplingGridVerticesScalar = NULL;

	threadsPerBlock = 32;
// 	gridDim.x = gridDim.y = gridDim.z = 1;
// 	blockDim.x = blockDim.y = blockDim.z = 1;

	deviceQuery();
}
MetaballCudaMgr::~MetaballCudaMgr()
{
//	cudaPrintfEnd();
}
MetaballCudaMgr* MetaballCudaMgr::getSingletonPtr()
{
	if(!m_singleton)
	{
		m_singleton = new MetaballCudaMgr();
	}
	return m_singleton;
}
//
void MetaballCudaMgr::mallocSpherePosition(const std::size_t elementCount, const std::size_t elementSize)
{
	assert( elementSize==sizeof(float)*3 );
	d_spherePositionsCount = elementCount;
	CUDA_SAFE_CALL( hipMalloc((void**)&d_spherePosition, elementCount*elementSize) );
}
void MetaballCudaMgr::setSpherePosition(const float *src, const std::size_t elementCount)
{
	assert(elementCount<=d_spherePositionsCount);
	CUDA_SAFE_CALL( hipMemcpy(d_spherePosition, src, d_spherePositionsCount*sizeof(float)*3, hipMemcpyHostToDevice) );
	d_spherePositionsCount = elementCount;
}
void MetaballCudaMgr::freeSpherePosition()
{
	if(d_spherePosition)
	{
		CUDA_SAFE_CALL( hipFree(d_spherePosition) );
		d_spherePositionsCount = 0;
	}
}
//
void MetaballCudaMgr::mallocSamplingGridVertices(const std::size_t elementCount, const std::size_t elementSize)
{
	assert( elementSize==sizeof(float)*4 );
	d_samplingGridVerticesCount = elementCount;
	CUDA_SAFE_CALL( hipMalloc((void**)&d_samplingGridVertices, elementCount*elementSize) );
	//CUDA_SAFE_CALL( hipMalloc((void**)&d_samplingGridVerticesScalar,   elementCount*sizeof(float)) );
}
void MetaballCudaMgr::setSamplingGridVertices(const float *src, const std::size_t elementCount)
{
	assert(elementCount<=d_samplingGridVerticesCount);
	CUDA_SAFE_CALL( hipMemcpy(d_samplingGridVertices, src, d_samplingGridVerticesCount*sizeof(float)*3, hipMemcpyHostToDevice) );
	d_samplingGridVerticesCount = elementCount;
}
void MetaballCudaMgr::freeSamplingGridVertices()
{
	if(d_samplingGridVertices)
	{
		CUDA_SAFE_CALL( hipFree(d_samplingGridVertices) );
		d_samplingGridVerticesCount = 0;
	}
}
//
void MetaballCudaMgr::mallocSamplingGridVerticesScalar(const std::size_t elementCount, const std::size_t elementSize)
{
	assert( elementSize==sizeof(float) );
	d_samplingGridVerticesScalarCount = elementCount;
	assert( d_samplingGridVerticesScalarCount == d_samplingGridVerticesCount);
	CUDA_SAFE_CALL( hipMalloc((void**)&d_samplingGridVerticesScalar, elementCount*elementSize) );
	CUDA_SAFE_CALL(	hipMemset(d_samplingGridVerticesScalar, 0, elementCount*elementSize) );
	
	h_samplingGridVerticesScalar = (float*)malloc(elementCount*elementSize);
	assert(h_samplingGridVerticesScalar);
}
// void MetaballCudaMgr::setSamplingGridVerticesScalar(const float *src, const std::size_t elementCount)
// {
// 	assert(elementCount<=d_samplingGridVerticesScalarCount);
// 	CUDA_SAFE_CALL( hipMemcpy(d_samplingGridVerticesScalar, src, d_samplingGridVerticesScalarCount*sizeof(float)*3, hipMemcpyHostToDevice) );
// 	d_samplingGridVerticesScalarCount = elementCount;
// 	assert( d_samplingGridVerticesScalarCount == d_samplingGridVerticesCount);
// }
void MetaballCudaMgr::freeSamplingGridVerticesScalar()
{
	if(d_samplingGridVerticesScalar)
	{
		CUDA_SAFE_CALL( hipFree(d_samplingGridVerticesScalar) );
		d_samplingGridVerticesScalarCount = 0;
		
		free(h_samplingGridVerticesScalar);
	}
}
void MetaballCudaMgr::Scalar()
{
	//launch_Scalar1D();
	launch_ScalarSphere1D_EffectRange();
	//launch_Scalar3D();
    CUDA_SAFE_CALL( hipMemcpy(
			h_samplingGridVerticesScalar, 
			d_samplingGridVerticesScalar, 
			d_samplingGridVerticesScalarCount*sizeof(float), 
			hipMemcpyDeviceToHost
		) 
	);
	CUDA_SAFE_CALL(	hipMemset(d_samplingGridVerticesScalar, 0, d_samplingGridVerticesScalarCount*sizeof(float)) );

}
void MetaballCudaMgr::launch_Scalar1D()
{
	//printf("MetaballCudaMgr::Scalar(%d, %d, %d)\n", gridSizeX, gridSizeY, gridSizeZ);
	//dim3 grid(gridSizeX, gridSizeY, gridSizeZ);
	//int threadsPerBlock = 32;
	//assign a thread for each grid vertex
    int blocksPerGrid = (d_samplingGridVerticesCount + threadsPerBlock - 1) / threadsPerBlock;
	//printf("launch_Scalar1D<<<%d, %d>>>\n", blocksPerGrid, threadsPerBlock);
	const float Threshold = 4.0f*sphereRadius;

	ScalarSphere1D<<<blocksPerGrid, threadsPerBlock>>>(
		d_spherePosition, 
		Threshold,
		d_samplingGridVertices, 
		d_samplingGridVerticesScalar,
		d_spherePositionsCount,
		d_samplingGridVerticesCount
	);
	//cudaPrintfDisplay(stdout, true);
    //CUDA_SAFE_CALL( hipDeviceSynchronize() );
}
void MetaballCudaMgr::launch_Scalar3D()
{
	//printf("MetaballCudaMgr::Scalar(%d, %d, %d)\n", gridSizeX, gridSizeY, gridSizeZ);
	//dim3 grid(gridSizeX, gridSizeY, gridSizeZ);
	//int threadsPerBlock = 32;
	//assign a thread for each grid vertex
    //int blocksPerGrid = (d_samplingGridVerticesCount + threadsPerBlock - 1) / threadsPerBlock;
	//printf("ScalarSphere<<<%d, %d>>>\n", blocksPerGrid, threadsPerBlock);

	dim3 gridDim(1,1,1);
	dim3 blockDim(ceil((float)d_samplingGridVerticesCount/(float)threadsPerBlock),1,1);
    while(blockDim.x > deviceProp.maxThreadsDim[0]) {
        blockDim.x/=2;
        blockDim.y*=2;
    }
	while(blockDim.y > deviceProp.maxThreadsDim[1]) {
        blockDim.y/=2;
        blockDim.z*=2;
    }
	while(blockDim.z> deviceProp.maxThreadsDim[2]){
		blockDim.z/=2;
		gridDim.x*=2;
	}
	while(gridDim.x> deviceProp.maxGridSize[0]){
		gridDim.x/=2;
		gridDim.y*=2;
	}
	while(gridDim.y> deviceProp.maxGridSize[1]){
		gridDim.y/=2;
		gridDim.z*=2;
	}
	//dim3 gridDim(1,1,1);
	//dim3 blockDim(582,1,1);
	printf("gridvertexcount=%d\n", d_samplingGridVerticesCount);
 	printf("gridDim<%u, %u, %u>\n", gridDim.x, gridDim.y, gridDim.z);
 	printf("blockDim<%u, %u, %u>\n", blockDim.x, blockDim.y, blockDim.z);
	assert(gridDim.z< deviceProp.maxGridSize[2]);

	//ScalarSphere3D<<<gridDim, blockDim, threadsPerBlock>>>(
	ScalarSphere3D<<<gridDim, blockDim, threadsPerBlock>>>(
		d_spherePosition, 
		d_samplingGridVertices, 
		d_samplingGridVerticesScalar,
		d_spherePositionsCount,
		d_samplingGridVerticesCount
	);
	//cudaPrintfDisplay(stdout, true);
    //CUDA_SAFE_CALL( hipDeviceSynchronize() );
}
void MetaballCudaMgr::FetchScalarValues(SamplingGridVertice *vertex, const std::size_t vertexCount)
{
	assert(vertexCount==d_samplingGridVerticesScalarCount);
	for(size_t i=0; i<vertexCount; ++i)
	{
        vertex[i].ScalarValue = h_samplingGridVerticesScalar[i];
	}

}
//
__global__ void ScalarSphere1D(const float* position, 
					   const float threshold,
					   const float* gridVertexPos, float* gridVertexScalar,
					   const int ParticleCount, const int GridVertexCount
					   )
{
	uint blockId = blockIdx.y*gridDim.x + blockIdx.x;
    uint vi = blockId*blockDim.x + threadIdx.x;
#if __CUDA_ARCH__ >= 200
	//printf("grid<%d, %d, %d>, ",                       gridDim.x,                  gridDim.y,                 gridDim.z);
	//printf("block<%d/%d, %d/%d, %d/%d>, ",blockIdx.x, blockDim.x,     blockIdx.y, blockDim.y,    blockIdx.z, blockDim.z);
	//printf("thread<%d, %d, %d>\n",       threadIdx.x,                threadIdx.y,               threadIdx.z            );
#endif	
	uint pi = 0;
	
	float scalarSum = 0.0f;
	
	if(vi < GridVertexCount)//gridVertexPos[vi]={gvx, gvy, gvz}
	{
		float gvx = gridVertexPos[4*vi  ];
		float gvy = gridVertexPos[4*vi+1];
		float gvz = gridVertexPos[4*vi+2];

		for(pi=0; pi<ParticleCount; ++pi)//position[pi]={px, py, pz}
		{
			float px = position[3*pi  ];
			float py = position[3*pi+1];
			float pz = position[3*pi+2];

			if( (px - gvx) >threshold || (px - gvx)<-threshold ) continue;
			if( (py - gvy) >threshold || (py - gvy)<-threshold ) continue;
			if( (pz - gvz) >threshold || (pz - gvz)<-threshold ) continue;

			scalarSum += sphereRadiusSquared /((px-gvx)*(px-gvx)+ 
												(py-gvy)*(py-gvy)+ 
												(pz-gvz)*(pz-gvz));
		}
	}
	gridVertexScalar[vi] = scalarSum;
}


__global__ void ScalarSphere3D(const float* position, 
					   const float* gridVertexPos, float* gridVertexScalar,
					   const int ParticleCount, const int GridVertexCount
					   )
{
	__shared__ uint threadIdxMax;

	uint gBlockID  =blockIdx.x +  
					blockIdx.y * gridDim.x + 
					blockIdx.z * gridDim.x * gridDim.y;

	uint gThreadID =threadIdx.x + 
					threadIdx.y * blockDim.x +
					threadIdx.z * blockDim.x * blockDim.y +
					gBlockID * blockDim.x * blockDim.y * blockDim.z;

    uint vi = gThreadID;
	threadIdxMax = max(threadIdxMax, gThreadID);
#if __CUDA_ARCH__ >= 200
	printf("threadIdxMax=%u\n", threadIdxMax);
	//printf("grid<%d, %d, %d>, ",                       gridDim.x,                  gridDim.y,                 gridDim.z);
	//printf("block<%d/%d, %d/%d, %d/%d>, ",blockIdx.x, blockDim.x,     blockIdx.y, blockDim.y,    blockIdx.z, blockDim.z);
	//printf("thread<%d, %d, %d>\n",       threadIdx.x,                threadIdx.y,               threadIdx.z            );
#endif	
	uint pi = 0;
	
	float scalarSum = 0.0f;
	
	if(vi < GridVertexCount)//gridVertexPos[vi]={gvx, gvy, gvz}
	{
		float gvx = gridVertexPos[4*vi  ];
		float gvy = gridVertexPos[4*vi+1];
		float gvz = gridVertexPos[4*vi+2];

		for(pi=0; pi<ParticleCount; ++pi)//position[pi]={px, py, pz}
		{
			float px = position[3*pi  ];
			float py = position[3*pi+1];
			float pz = position[3*pi+2];

			scalarSum += sphereRadiusSquared /((px-gvx)*(px-gvx)+ 
												(py-gvy)*(py-gvy)+ 
												(pz-gvz)*(pz-gvz));
		}
	}
	gridVertexScalar[vi] = scalarSum;
}
//
void MetaballCudaMgr::SetThreadsPerBlock(const std::size_t threadsPerBlock_)
{
	assert(threadsPerBlock_<= deviceProp.maxThreadsPerBlock);
	threadsPerBlock = threadsPerBlock_;
}
//  void MetaballCudaMgr::SetGridDim( const std::size_t gridx,
//  								  const std::size_t gridy,
//  								  const std::size_t gridz)
//  {
//  	gridDim.x = gridx * gridy * gridz;
//      while(gridDim.x > deviceProp.maxGridSize[0]) {//
//          gridDim.x/=2;
//          gridDim.y*=2;
//      }
//  	while(gridDim.y > deviceProp.maxGridSize[1]) {//
//          gridDim.y/=2;
//          gridDim.z*=2;
//      }
//  	assert(gridDim.z < deviceProp.maxGridSize[2]);
//  	printf("gridDim<%d, %d, %d>\n", gridDim.x, gridDim.y, gridDim.z);
//  }
//  void MetaballCudaMgr::SetBlockDim(const std::size_t blockx,
//  								  const std::size_t blocky,
//  								  const std::size_t blockz)
//  {
//  	blockDim.x = blockx * blocky * blockz;
//      while(blockDim.x > deviceProp.maxThreadsDim[0]) {
//          blockDim.x/=2;
//          blockDim.y*=2;
//      }
//  	while(blockDim.y > deviceProp.maxThreadsDim[1]) {
//          blockDim.y/=2;
//          blockDim.z*=2;
//      }
//  	assert(blockDim.z< deviceProp.maxThreadsDim[2]);
//  	printf("blockDim<%d, %d, %d>\n", blockDim.x, blockDim.y, blockDim.z);
//  }
void MetaballCudaMgr::deviceQuery()
{
	int deviceCount = 0;
	if (hipGetDeviceCount(&deviceCount) != hipSuccess) {
		printf("hipGetDeviceCount FAILED CUDA Driver and Runtime version may be mismatched.\n");
		printf("\nFAILED\n");
	}
	if (deviceCount == 0)
        printf("There is no device supporting CUDA\n");

    hipGetDeviceProperties(&deviceProp, 0);//device0

}
void MetaballCudaMgr::launch_ScalarSphere1D_EffectRange()
{
	//printf("MetaballCudaMgr::Scalar(%d, %d, %d)\n", gridSizeX, gridSizeY, gridSizeZ);
	//dim3 grid(gridSizeX, gridSizeY, gridSizeZ);
	//int threadsPerBlock = 32;
	//assign a thread for each grid vertex
    int blocksPerGrid = (d_spherePositionsCount + threadsPerBlock - 1) / threadsPerBlock;
	//printf("launch_Scalar1D<<<%d, %d>>>\n", blocksPerGrid, threadsPerBlock);
//	const float Threshold = 4.0f*sphereRadius;

	ScalarSphere1D_EffectRange<<<blocksPerGrid, threadsPerBlock>>>(
		d_spherePosition,
		d_samplingGridVertices, 
		d_samplingGridVerticesScalar,
		d_spherePositionsCount,
		d_samplingGridVerticesCount
	);
	//cudaPrintfDisplay(stdout, true);
    //CUDA_SAFE_CALL( hipDeviceSynchronize() );
}
long int __device__ getVertexIdx(uint x, uint y, uint z)
{
	return (x + y * NBRSamples.x + z * NBRSamples.x * NBRSamples.y);
}
__global__ void ScalarSphere1D_EffectRange(
					   const float* position, 
					   const float* gridVertexPos, float* gridVertexScalar,
					   const int ParticleCount, const int GridVertexCount
					   )
{
	uint blockId = blockIdx.y*gridDim.x + blockIdx.x;
    uint pi = blockId*blockDim.x + threadIdx.x;
#if __CUDA_ARCH__ >= 200
	//printf("grid<%d, %d, %d>, ",                       gridDim.x,                  gridDim.y,                 gridDim.z);
	//printf("block<%d/%d, %d/%d, %d/%d>, ",blockIdx.x, blockDim.x,     blockIdx.y, blockDim.y,    blockIdx.z, blockDim.z);
	//printf("thread<%d, %d, %d>\n",       threadIdx.x,                threadIdx.y,               threadIdx.z            );
#endif	
	if( pi< ParticleCount )
	{
		dim3 gridCubeIdx;
		gridCubeIdx.x = ceil(position[3*pi  ] / SpaceResolution.x);
		gridCubeIdx.y = ceil(position[3*pi+1] / SpaceResolution.y);
		gridCubeIdx.z = ceil(position[3*pi+2] / SpaceResolution.z);

		dim3 min_, max_;
		min_.x = fmaxf(0, gridCubeIdx.x-ExtendCubes.x);
		min_.y = fmaxf(0, gridCubeIdx.y-ExtendCubes.y);
		min_.z = fmaxf(0, gridCubeIdx.z-ExtendCubes.z);
		max_.x = fminf(gridCubeIdx.x+ExtendCubes.x, NBRSamples.x);
		max_.y = fminf(gridCubeIdx.y+ExtendCubes.y, NBRSamples.y);
		max_.z = fminf(gridCubeIdx.z+ExtendCubes.z, NBRSamples.z);

		for(uint k = min_.z; k <max_.z; ++k)
		{
 			for(uint j = min_.y; j <max_.y; ++j)
 			{
 				for(uint i = min_.x; i <max_.x; ++i)
 				{
 					long int vIdx = getVertexIdx(i, j, k);
#if __CUDA_ARCH__ >= 200
 					uint tmp = vIdx;
					if(vIdx != tmp)
					{

						printf("over flow vIdx=%ll", vIdx);

					}
#endif
					if(vIdx<0 || vIdx>GridVertexCount)
						continue;

					float gvx = gridVertexPos[4*vIdx  ];
					float gvy = gridVertexPos[4*vIdx+1];
					float gvz = gridVertexPos[4*vIdx+2];

					float px = position[3*pi  ];
					float py = position[3*pi+1];
					float pz = position[3*pi+2];
					
					gridVertexScalar[vIdx] += 
						sphereRadiusSquared /((px-gvx)*(px-gvx)+ (py-gvy)*(py-gvy)+ (pz-gvz)*(pz-gvz));
 				}//for
 			}//for 
 		}//for 
	}
}
void MetaballCudaMgr::SetSamplesNumber(const unsigned int smplx, const unsigned int smply, const unsigned int smplz)
{
	uint3 tmp;
	tmp.x = smplx;
	tmp.y = smply;
	tmp.z = smplz;
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(NBRSamples), &tmp, sizeof(uint3)) );
}
void MetaballCudaMgr::SetSpaceResolution(const float resx, const float resy, const float resz)
{
	float3 tmp;
	tmp.x = resx;
	tmp.y = resy;
	tmp.z = resz;
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(SpaceResolution), &tmp, sizeof(float3)) );
}
void MetaballCudaMgr::SetExtendCubes(const unsigned int N, const float sphereRadius_,
	const float resx, const float resy, const float resz)
{
	dim3 tmp(N*sphereRadius_/resx, N*sphereRadius_/resy, N*sphereRadius_/resz);
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(ExtendCubes), &tmp, sizeof(dim3)) );
}
void MetaballCudaMgr::initilize(
	const unsigned int NBRSamplesX, const unsigned int NBRSamplesY, const unsigned int NBRSamplesZ,
	const float resx, const float resy, const float resz,
	const unsigned int N,
	const float sphereRadius_
)
{
	SetSamplesNumber(NBRSamplesX, NBRSamplesY, NBRSamplesZ);
	SetSpaceResolution(resx, resy, resz);
	   
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(sphereRadius), &sphereRadius_, sizeof(float)) );
	const float tmp(sphereRadius_*sphereRadius_);
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(sphereRadiusSquared), &tmp, sizeof(float)) );

	SetExtendCubes(N, sphereRadius_, resx, resy, resz);
}
