#include "hip/hip_runtime.h"
#include "MetaballCudaMgr.cuh"

#include <cassert>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include "./trunk/OgreMetaballs/MarchingCubesInterface.h"
//#include "../SPHSimLib/cuPrintf.cu"

__global__ void ScalarSphere1D(const float* position, 
					   const float radiusSquared,
					   const float* gridVertex, float* scalar,
					   const int ParticleCount, const int GridVertexCount);
// __global__ void ScalarSphere3D(const float* position, 
// 					   const float radiusSquared,
// 					   const float* gridVertex, float* scalar,
// 					   const int ParticleCount, const int GridVertexCount);

MetaballCudaMgr* MetaballCudaMgr::m_singleton=NULL;

MetaballCudaMgr::MetaballCudaMgr()
{
//	cudaPrintfInit();
	d_spherePosition = NULL;
	d_spherePositionsCount = 0;
	sphereRadius = 1.0;

	d_samplingGridVertices = NULL;
	d_samplingGridVerticesCount = 0;
	
	d_samplingGridVerticesScalar = NULL;
	d_samplingGridVerticesScalarCount = 0;
	h_samplingGridVerticesScalar = NULL;

	threadsPerBlock = 32;
	//gridDim.x = gridDim.y = gridDim.z = 1;
	//blockDim.x = blockDim.y = blockDim.z = 1;
	
	deviceQuery();
}
MetaballCudaMgr::~MetaballCudaMgr()
{
//	cudaPrintfEnd();
}
MetaballCudaMgr* MetaballCudaMgr::getSingletonPtr()
{
	if(!m_singleton)
	{
		m_singleton = new MetaballCudaMgr();
	}
	return m_singleton;
}
//
void MetaballCudaMgr::mallocSpherePosition(const std::size_t elementCount, const std::size_t elementSize)
{
	assert( elementSize==sizeof(float)*3 );
	d_spherePositionsCount = elementCount;
	CUDA_SAFE_CALL( hipMalloc((void**)&d_spherePosition, elementCount*elementSize) );
}
void MetaballCudaMgr::setSpherePosition(const float *src, const std::size_t elementCount)
{
	assert(elementCount<=d_spherePositionsCount);
	CUDA_SAFE_CALL( hipMemcpy(d_spherePosition, src, d_spherePositionsCount*sizeof(float)*3, hipMemcpyHostToDevice) );
	d_spherePositionsCount = elementCount;
}
void MetaballCudaMgr::freeSpherePosition()
{
	if(d_spherePosition)
	{
		CUDA_SAFE_CALL( hipFree(d_spherePosition) );
		d_spherePositionsCount = 0;
	}
}
//
void MetaballCudaMgr::mallocSamplingGridVertices(const std::size_t elementCount, const std::size_t elementSize)
{
	assert( elementSize==sizeof(float)*4 );
	d_samplingGridVerticesCount = elementCount;
	CUDA_SAFE_CALL( hipMalloc((void**)&d_samplingGridVertices, elementCount*elementSize) );
	//CUDA_SAFE_CALL( hipMalloc((void**)&d_samplingGridVerticesScalar,   elementCount*sizeof(float)) );
}
void MetaballCudaMgr::setSamplingGridVertices(const float *src, const std::size_t elementCount)
{
	assert(elementCount<=d_samplingGridVerticesCount);
	CUDA_SAFE_CALL( hipMemcpy(d_samplingGridVertices, src, d_samplingGridVerticesCount*sizeof(float)*3, hipMemcpyHostToDevice) );
	d_samplingGridVerticesCount = elementCount;
}
void MetaballCudaMgr::freeSamplingGridVertices()
{
	if(d_samplingGridVertices)
	{
		CUDA_SAFE_CALL( hipFree(d_samplingGridVertices) );
		d_samplingGridVerticesCount = 0;
	}
}
//
void MetaballCudaMgr::mallocSamplingGridVerticesScalar(const std::size_t elementCount, const std::size_t elementSize)
{
	assert( elementSize==sizeof(float) );
	d_samplingGridVerticesScalarCount = elementCount;
	assert( d_samplingGridVerticesScalarCount == d_samplingGridVerticesCount);
	CUDA_SAFE_CALL( hipMalloc((void**)&d_samplingGridVerticesScalar, elementCount*elementSize) );
	
	h_samplingGridVerticesScalar = (float*)malloc(elementCount*elementSize);
	assert(h_samplingGridVerticesScalar);
}
// void MetaballCudaMgr::setSamplingGridVerticesScalar(const float *src, const std::size_t elementCount)
// {
// 	assert(elementCount<=d_samplingGridVerticesScalarCount);
// 	CUDA_SAFE_CALL( hipMemcpy(d_samplingGridVerticesScalar, src, d_samplingGridVerticesScalarCount*sizeof(float)*3, hipMemcpyHostToDevice) );
// 	d_samplingGridVerticesScalarCount = elementCount;
// 	assert( d_samplingGridVerticesScalarCount == d_samplingGridVerticesCount);
// }
void MetaballCudaMgr::freeSamplingGridVerticesScalar()
{
	if(d_samplingGridVerticesScalar)
	{
		CUDA_SAFE_CALL( hipFree(d_samplingGridVerticesScalar) );
		d_samplingGridVerticesScalarCount = 0;
		
		free(h_samplingGridVerticesScalar);
	}
}
void MetaballCudaMgr::Scalar()
{
	launch_Scalar1D();
	//launch_Scalar3D();
    CUDA_SAFE_CALL( hipMemcpy(
			h_samplingGridVerticesScalar, 
			d_samplingGridVerticesScalar, 
			d_samplingGridVerticesScalarCount*sizeof(float), 
			hipMemcpyDeviceToHost
		) 
	);
}
void MetaballCudaMgr::launch_Scalar1D()
{
	//printf("MetaballCudaMgr::Scalar(%d, %d, %d)\n", gridSizeX, gridSizeY, gridSizeZ);
	//dim3 grid(gridSizeX, gridSizeY, gridSizeZ);
	//int threadsPerBlock = 32;
	//assign a thread for each grid vertex
    int blocksPerGrid = (d_samplingGridVerticesCount + threadsPerBlock - 1) / threadsPerBlock;
	//printf("launch_Scalar1D<<<%d, %d>>>\n", blocksPerGrid, threadsPerBlock);

	ScalarSphere1D<<<blocksPerGrid, threadsPerBlock>>>(
		d_spherePosition, 
		sphereRadius*sphereRadius,
		d_samplingGridVertices, 
		d_samplingGridVerticesScalar,
		d_spherePositionsCount,
		d_samplingGridVerticesCount
	);
	//cudaPrintfDisplay(stdout, true);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
}
// void MetaballCudaMgr::launch_Scalar3D()
// {
// 	//printf("MetaballCudaMgr::Scalar(%d, %d, %d)\n", gridSizeX, gridSizeY, gridSizeZ);
// 	//dim3 grid(gridSizeX, gridSizeY, gridSizeZ);
// 	//int threadsPerBlock = 32;
// 	//assign a thread for each grid vertex
//     //int blocksPerGrid = (d_samplingGridVerticesCount + threadsPerBlock - 1) / threadsPerBlock;
// 	//printf("ScalarSphere<<<%d, %d>>>\n", blocksPerGrid, threadsPerBlock);
// 	dim3 gridDim(1,1,1);
	dim3 blockDim(d_samplingGridVerticesCount,1,1);
// 
//     while(blockDim.x > deviceProp.maxThreadsDim[0]) {
//         blockDim.x/=2;
//         blockDim.y*=2;
//     }
// 	while(blockDim.y > deviceProp.maxThreadsDim[1]) {
//         blockDim.y/=2;
//         blockDim.z*=2;
//     }
// 	while(blockDim.z> deviceProp.maxThreadsDim[2]){
// 		blockDim.z/=2;
// 		gridDim.x*=2;
// 	}
// 	while(gridDim.x> deviceProp.maxGridSize[0]){
// 		gridDim.x/=2;
// 		gridDim.y*=2;
// 	}
// 	while(gridDim.y> deviceProp.maxGridSize[1]){
// 		gridDim.y/=2;
// 		gridDim.z*=2;
// 	}
//  	printf("gridDim<%u, %u, %u>\n", gridDim.x, gridDim.y, gridDim.z);
//  	printf("blockDim<%u, %u, %u>\n", blockDim.x, blockDim.y, blockDim.z);
// 	assert(gridDim.z< deviceProp.maxGridSize[2]);
// 
// 	ScalarSphere3D<<<gridDim, blockDim, threadsPerBlock>>>(
// 		d_spherePosition, 
// 		sphereRadius*sphereRadius,
// 		d_samplingGridVertices, 
// 		d_samplingGridVerticesScalar,
// 		d_spherePositionsCount,
// 		d_samplingGridVerticesCount
// 	);
// 	//cudaPrintfDisplay(stdout, true);
//     //CUDA_SAFE_CALL( hipDeviceSynchronize() );
// }
void MetaballCudaMgr::FetchScalarValues(SamplingGridVertice *vertex, const std::size_t vertexCount)
{
	assert(vertexCount==d_samplingGridVerticesScalarCount);
	for(size_t i=0; i<vertexCount; ++i)
	{
        vertex[i].ScalarValue = h_samplingGridVerticesScalar[i];
	}
}
//
__global__ void ScalarSphere1D(const float* position, 
					   const float radiusSquared,
					   const float* gridVertexPos, float* gridVertexScalar,
					   const int ParticleCount, const int GridVertexCount
					   )
{
	uint blockId = blockIdx.y*gridDim.x + blockIdx.x;
    uint vi = blockId*blockDim.x + threadIdx.x;
#if __CUDA_ARCH__ >= 200
	//printf("grid<%d, %d, %d>, ",                       gridDim.x,                  gridDim.y,                 gridDim.z);
	//printf("block<%d/%d, %d/%d, %d/%d>, ",blockIdx.x, blockDim.x,     blockIdx.y, blockDim.y,    blockIdx.z, blockDim.z);
	//printf("thread<%d, %d, %d>\n",       threadIdx.x,                threadIdx.y,               threadIdx.z            );
#endif	
	uint pi = 0;
	
	float scalarSum = 0.0f;
	
	if(vi < GridVertexCount)//gridVertexPos[vi]={gvx, gvy, gvz}
	{
		float gvx = gridVertexPos[4*vi  ];
		float gvy = gridVertexPos[4*vi+1];
		float gvz = gridVertexPos[4*vi+2];

		for(pi=0; pi<ParticleCount; ++pi)//position[pi]={px, py, pz}
		{
			float px = position[3*pi  ];
			float py = position[3*pi+1];
			float pz = position[3*pi+2];

			scalarSum += radiusSquared /((px-gvx)*(px-gvx)+ 
										(py-gvy)*(py-gvy)+ 
										(pz-gvz)*(pz-gvz));
		}
	}
	gridVertexScalar[vi] = scalarSum;
}
// __global__ void ScalarSphere3D(const float* position, 
// 					   const float radiusSquared,
// 					   const float* gridVertexPos, float* gridVertexScalar,
// 					   const int ParticleCount, const int GridVertexCount
// 					   )
// {
// 	uint gBlockID  =blockIdx.x +  
					blockIdx.y * gridDim.x + 
					blockIdx.z * gridDim.x * gridDim.y;

	uint gThreadID =threadIdx.x + 
					threadIdx.y * blockDim.x +
					threadIdx.z * blockDim.x * blockDim.y +
					gBlockID * blockDim.x * blockDim.y * blockDim.z;
// 
//     uint vi = gThreadID;
// #if __CUDA_ARCH__ >= 200
// 	//printf("grid<%d, %d, %d>, ",                       gridDim.x,                  gridDim.y,                 gridDim.z);
// 	//printf("block<%d/%d, %d/%d, %d/%d>, ",blockIdx.x, blockDim.x,     blockIdx.y, blockDim.y,    blockIdx.z, blockDim.z);
// 	//printf("thread<%d, %d, %d>\n",       threadIdx.x,                threadIdx.y,               threadIdx.z            );
// #endif	
// 	uint pi = 0;
// 	
// 	float scalarSum = 0.0f;
// 	
// 	if(vi < GridVertexCount)//gridVertexPos[vi]={gvx, gvy, gvz}
// 	{
// 		float gvx = gridVertexPos[4*vi  ];
// 		float gvy = gridVertexPos[4*vi+1];
// 		float gvz = gridVertexPos[4*vi+2];
// 
// 		for(pi=0; pi<ParticleCount; ++pi)//position[pi]={px, py, pz}
// 		{
// 			float px = position[3*pi  ];
// 			float py = position[3*pi+1];
// 			float pz = position[3*pi+2];
// 
// 			scalarSum += radiusSquared /((px-gvx)*(px-gvx)+ 
// 										(py-gvy)*(py-gvy)+ 
// 										(pz-gvz)*(pz-gvz));
// 		}
// 	}
// 	gridVertexScalar[vi] = scalarSum;
// }
//
void MetaballCudaMgr::SetThreadsPerBlock(const std::size_t threadsPerBlock_)
{
	assert(threadsPerBlock_< deviceProp.maxThreadsPerBlock);
	threadsPerBlock = threadsPerBlock_;
}
// void MetaballCudaMgr::SetGridDim( const std::size_t gridx,
// 								  const std::size_t gridy,
// 								  const std::size_t gridz)
// {
// 	gridDim.x = gridx * gridy * gridz;
//     while(gridDim.x > deviceProp.maxGridSize[0]) {//
//         gridDim.x/=2;
//         gridDim.y*=2;
//     }
// 	while(gridDim.y > deviceProp.maxGridSize[1]) {//
//         gridDim.y/=2;
//         gridDim.z*=2;
//     }
// 	assert(gridDim.z < deviceProp.maxGridSize[2]);
// 	printf("gridDim<%d, %d, %d>\n", gridDim.x, gridDim.y, gridDim.z);
// }
// void MetaballCudaMgr::SetBlockDim(const std::size_t blockx,
// 								  const std::size_t blocky,
// 								  const std::size_t blockz)
// {
// 	blockDim.x = blockx * blocky * blockz;
//     while(blockDim.x > deviceProp.maxThreadsDim[0]) {
//         blockDim.x/=2;
//         blockDim.y*=2;
//     }
// 	while(blockDim.y > deviceProp.maxThreadsDim[1]) {
//         blockDim.y/=2;
//         blockDim.z*=2;
//     }
// 	assert(blockDim.z< deviceProp.maxThreadsDim[2]);
// 	printf("blockDim<%d, %d, %d>\n", blockDim.x, blockDim.y, blockDim.z);
// }
void MetaballCudaMgr::deviceQuery()
{
	int deviceCount = 0;
	if (hipGetDeviceCount(&deviceCount) != hipSuccess) {
		printf("hipGetDeviceCount FAILED CUDA Driver and Runtime version may be mismatched.\n");
		printf("\nFAILED\n");
	}
	if (deviceCount == 0)
        printf("There is no device supporting CUDA\n");

    hipGetDeviceProperties(&deviceProp, 0);//device0

}